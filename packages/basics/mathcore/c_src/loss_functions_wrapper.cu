#include "hip/hip_runtime.h"
/*
 * This file is part of the Neural Network modules of the APRIL toolkit (A
 * Pattern Recognizer In Lua).
 *
 * Copyright 2012, Salvador España-Boquera, Adrian Palacios Corella, Francisco
 * Zamora-Martinez
 *
 * The APRIL-ANN toolkit is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License version 3 as
 * published by the Free Software Foundation
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License
 * for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this library; if not, write to the Free Software Foundation,
 * Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA
 *
 */
#include <cmath>
#include "clamp.h"
#include "error_print.h"
#include "wrapper.h"
#include "unused_variable.h"
#include "april_assert.h"

using april_utils::clamp;

#define clip(value, min, max) (((value) < (min)) ? (min) : (((value) > (max)) ? (max) : (value)))

///////////////////////////////////////////////////////////
/////////////////// Kernels ///////////////////////////////
///////////////////////////////////////////////////////////

#ifdef USE_CUDA
#include "cuda_utils.h"
__global__ void computeMSELossFunctionKernel(const float *output,
					     const float *target_output,
					     float *pattern_errors,
					     float zero_epsilon_distance,
					     unsigned int max_x,
					     unsigned int lda_x,
					     unsigned int max_y) {
  unsigned int matrix_x_pos, matrix_y_pos;
  getColumnMajorBunchMatrixPositions(blockIdx,
				     blockDim,
				     threadIdx,
				     matrix_x_pos,
				     matrix_y_pos);
  if (matrix_x_pos < max_x && matrix_y_pos < max_y) {
    unsigned int index = getMatrixFlatIndex(matrix_x_pos, lda_x, matrix_y_pos);
    float d = output[index] - target_output[index];
    if (fabsf(d) < zero_epsilon_distance) d = 0.0f;
    pattern_errors[index] = d*d;
  }
}

__global__ void computeMSEGradientKernel(const float *output,
					 const float *target_output,
					 float *error_output,
					 float zero_epsilon_distance,
					 unsigned int max_x,
					 unsigned int lda_x,
					 unsigned int max_y) {
  unsigned int matrix_x_pos, matrix_y_pos;
  getColumnMajorBunchMatrixPositions(blockIdx,
				     blockDim,
				     threadIdx,
				     matrix_x_pos,
				     matrix_y_pos);
  if (matrix_x_pos < max_x && matrix_y_pos < max_y) {
    unsigned int index = getMatrixFlatIndex(matrix_x_pos, lda_x, matrix_y_pos);
    float d = output[index] - target_output[index];
    if (fabsf(d) < zero_epsilon_distance) d = 0.0f;
    error_output[index] = d;
  }
}

__global__ void computeMAELossFunctionKernel(const float *output,
					     const float *target_output,
					     float *pattern_errors,
					     float zero_epsilon_distance,
					     unsigned int max_x,
					     unsigned int lda_x,
					     unsigned int max_y) {
  unsigned int matrix_x_pos, matrix_y_pos;
  getColumnMajorBunchMatrixPositions(blockIdx,
				     blockDim,
				     threadIdx,
				     matrix_x_pos,
				     matrix_y_pos);
  if (matrix_x_pos < max_x && matrix_y_pos < max_y) {
    unsigned int index = getMatrixFlatIndex(matrix_x_pos, lda_x, matrix_y_pos);
    float absd = fabsf(output[index] - target_output[index]);
    if (absd < zero_epsilon_distance) absd = 0.0f;
    pattern_errors[index] = absd / max_y;
  }
}

__global__ void computeMAEGradientKernel(const float *output,
					 const float *target_output,
					 float *error_output,
					 float zero_epsilon_distance,
					 unsigned int max_x,
					 unsigned int lda_x,
					 unsigned int max_y,
					 float invN) {
  unsigned int matrix_x_pos, matrix_y_pos;
  getColumnMajorBunchMatrixPositions(blockIdx,
				     blockDim,
				     threadIdx,
				     matrix_x_pos,
				     matrix_y_pos);
  if (matrix_x_pos < max_x && matrix_y_pos < max_y) {
    unsigned int index = getMatrixFlatIndex(matrix_x_pos, lda_x, matrix_y_pos);
    float d = output[index] - target_output[index];
    if (fabsf(d) < zero_epsilon_distance) error_output[index] = 0.0f;
    else {
      if (d < 0.0f) error_output[index] = -invN;
      else error_output[index] = invN;
    }
  }
}

__global__ void computeMultiClassCrossEntropyLossFunctionKernel(const float *output,
								const float *target_output,
								float *pattern_errors,
								float epsilon,
								unsigned int max_x,
								unsigned int lda_x,
								unsigned int max_y) {
  unsigned int matrix_x_pos, matrix_y_pos;
  getColumnMajorBunchMatrixPositions(blockIdx,
				     blockDim,
				     threadIdx,
				     matrix_x_pos,
				     matrix_y_pos);
  if (matrix_x_pos < max_x && matrix_y_pos < max_y) {
    unsigned int index = getMatrixFlatIndex(matrix_x_pos, lda_x, matrix_y_pos);
    // compute derivative
    // float o = clip(output[index], inf, epsilon, 1.0f - epsilon);
    float log_o = output[index];
    float t = clip(target_output[index], epsilon, 1.0f - epsilon);
    if (t > epsilon) pattern_errors[index] = t * log_o;
    else pattern_errors[index] = 0.0f;
  }
}

__global__ void computeCrossEntropyLossFunctionKernel(const float *output,
						      const float *target_output,
						      float *pattern_errors,
						      float epsilon,
						      unsigned int max_x,
						      unsigned int lda_x,
						      unsigned int max_y) {
  unsigned int matrix_x_pos, matrix_y_pos;
  getColumnMajorBunchMatrixPositions(blockIdx,
				     blockDim,
				     threadIdx,
				     matrix_x_pos,
				     matrix_y_pos);
  if (matrix_x_pos < max_x && matrix_y_pos < max_y) {
    unsigned int index = getMatrixFlatIndex(matrix_x_pos, lda_x, matrix_y_pos);
    // compute derivative
    float  log_o     = clip(output[index], logf(epsilon), logf(1.0 - epsilon));
    double o         = exp(output[index]);
    float  log_inv_o = log(1.0 - o);
    float  t         = clip(target_output[index], epsilon, 1.0f - epsilon);
    float  inv_t     = clip(1.0f - target_output[index], epsilon, 1.0f - epsilon);
    if (t > epsilon) pattern_errors[index] = t * log_o;
    else pattern_errors[index] = 0.0f;
    if (inv_t > epsilon) pattern_errors[index] += inv_t * log_inv_o;
  }
}

__global__ void computeCrossEntropyGradientKernel(const float *output,
						  const float *target_output,
						  float *error_output,
						  float zero,
						  unsigned int max_x,
						  unsigned int lda_x,
						  unsigned int max_y) {
  unsigned int matrix_x_pos, matrix_y_pos;
  getColumnMajorBunchMatrixPositions(blockIdx,
				     blockDim,
				     threadIdx,
				     matrix_x_pos,
				     matrix_y_pos);
  if (matrix_x_pos < max_x && matrix_y_pos < max_y) {
    unsigned int index = getMatrixFlatIndex(matrix_x_pos, lda_x, matrix_y_pos);
    // compute derivative
    error_output[index] = expf(output[index]) - target_output[index];
  }
}

__global__ void applyTanhErrorFunctionKernel(const float *output,
					     const float *target_output,
					     float *output_error,
					     float *pattern_errors,
					     unsigned int max_x,
					     unsigned int lda_x,
					     unsigned int max_y) {
  unsigned int matrix_x_pos, matrix_y_pos;
  getColumnMajorBunchMatrixPositions(blockIdx,
				     blockDim,
				     threadIdx,
				     matrix_x_pos,
				     matrix_y_pos);
  if (matrix_x_pos < max_x && matrix_y_pos < max_y) {
    unsigned int index = getMatrixFlatIndex(matrix_x_pos, lda_x, matrix_y_pos);
    float d = output_error[index] = output[index] - target_output[index];
    if (d < -0.9999999f)
      output_error[index] = -DERIVATIVE_SATURATION;
    else if (d > 0.9999999f)
      output_error[index] =  DERIVATIVE_SATURATION;
    else output_error[index] = log((1.0f+output_error[index])/(1.0f-output_error[index]));
    pattern_errors[index] = d*d;
  }
}

void sumBunchPatternErrors(float *loss_output_ptr,
			   const float *pattern_errors_ptr,
			   unsigned int bunch_size, unsigned int size) {
  hipblasHandle_t handle  = GPUHelper::getHandler();
  int aux_size = static_cast<int>(size);
  for (unsigned int i=0; i<bunch_size; ++i)
    hipblasSasum(handle, aux_size,
		pattern_errors_ptr+i, bunch_size,
		loss_output_ptr + i);
}

#endif


///////////////////////////////////////////////////////////
///////////////// Error functions wrappers ////////////////
///////////////////////////////////////////////////////////

/// Generic macro for expansion of loss functions code which are computed as a
/// sum of the loss for every position of input/target matrices. The sum is
/// computed in two loops, and traversing in col-major (first by bunch_size).
/// The first loop computes the loss of the first component of all the patterns,
/// and stores it at the loss_output_ptr. The second loop computes the loss of
/// the rest of components, adding it to the loss_output_ptr.
/// The parameters of the macro are:
/// @param[in]  input   A FloatGPUMirroredMemoryBlock pointer.
/// @param[in]  target  A FloatGPUMirroredMemoryBlock pointer.
/// @param[out] loss_output  A FloatGPUMirroredMemoryBlock pointer.
/// @param[in]  size  The number of components in one pattern.
/// @param[in]  bunch_size  The number of patterns.
/// @param[in]  CODE  The code which will be executed.
/// @param[in]  var  The variable where the CODE stores the loss.
#define COMPUTE_LOSS(input,target,loss_output,size,bunch_size,CODE,var)	\
  do {									\
    const float *input_ptr  = (input)->getPPALForRead();		\
    const float *target_ptr = (target)->getPPALForRead();		\
    float *loss_output_ptr  = (loss_output)->getPPALForWrite();		\
    for (unsigned int b=0; b<(bunch_size); ++b) {			\
      CODE;								\
      loss_output_ptr[b] = (var);					\
    }									\
    input_ptr  += bunch_size;						\
    target_ptr += bunch_size;						\
    for (unsigned int i = 1; i < (size); i++) {				\
      for (unsigned int b=0; b<(bunch_size); ++b) {			\
	CODE;								\
	loss_output_ptr[b] += (var);					\
      }									\
      input_ptr  += bunch_size;						\
      target_ptr += bunch_size;						\
    }									\
  } while(0)

void doMSELossFunction(FloatGPUMirroredMemoryBlock *input,
		       FloatGPUMirroredMemoryBlock *target,
		       FloatGPUMirroredMemoryBlock *loss_output,
		       float zero_epsilon_distance,
		       unsigned int size,
		       unsigned int bunch_size,
		       bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {    
    const float *input_ptr  = input->getGPUForRead();
    const float *target_ptr = target->getGPUForRead();
    FloatGPUMirroredMemoryBlock *pattern_errors = 
      new FloatGPUMirroredMemoryBlock(target->getSize());
    float *pattern_errors_ptr = pattern_errors->getGPUForWrite();
    dim3 block, grid;
    computeBlockAndGridSizesForAColumnMajorBunch(bunch_size, size,
						 block, grid);
    computeMSELossFunctionKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (input_ptr,
       target_ptr,
       pattern_errors_ptr,
       zero_epsilon_distance,
       bunch_size,
       bunch_size,
       size);
    sumBunchPatternErrors(loss_output->getGPUForWrite(),
			  pattern_errors_ptr,
			  bunch_size, size);
    delete pattern_errors;
  }
  else {
#endif
    COMPUTE_LOSS(input,target,loss_output,size,bunch_size,
		 float d = input_ptr[b] - target_ptr[b];
		 if (fabs(d) < zero_epsilon_distance) d = 0.0f;
		 else d = d*d,
		 d);
#ifdef USE_CUDA
  }
#endif
}


void doComputeMSEGradient(FloatGPUMirroredMemoryBlock *input,
			  FloatGPUMirroredMemoryBlock *target,
			  FloatGPUMirroredMemoryBlock *error_output,
			  float zero_epsilon_distance,
			  unsigned int size,
			  unsigned int bunch_size,
			  bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {    
    const float *input_ptr  = input->getGPUForRead();
    const float *target_ptr = target->getGPUForRead();
    float *error_output_ptr = error_output->getGPUForWrite();
    dim3 block, grid;
    computeBlockAndGridSizesForAColumnMajorBunch(bunch_size, size,
						 block, grid);
    computeMSEGradientKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (input_ptr,
       target_ptr,
       error_output_ptr,
       zero_epsilon_distance,
       bunch_size,
       bunch_size,
       size);
  }
  else {
#endif
    float d = 0.0f;
    const float *input_ptr  = input->getPPALForRead();
    const float *target_ptr = target->getPPALForRead();
    float *error_output_ptr = error_output->getPPALForWrite();
    for (unsigned int i = 0; i < size; i++) {
      for (unsigned int b=0; b<bunch_size; ++b) {
	d = input_ptr[b] - target_ptr[b];
	if (fabsf(d) < zero_epsilon_distance) d = 0.0f;
	error_output_ptr[b] = d;
      }
      input_ptr  += bunch_size;
      target_ptr += bunch_size;
      error_output_ptr += bunch_size;
    }
#ifdef USE_CUDA
  }
#endif
}

void doMAELossFunction(FloatGPUMirroredMemoryBlock *input,
		       FloatGPUMirroredMemoryBlock *target,
		       FloatGPUMirroredMemoryBlock *loss_output,
		       float zero_epsilon_distance,
		       unsigned int size,
		       unsigned int bunch_size,
		       bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {    
    const float *input_ptr  = input->getGPUForRead();
    const float *target_ptr = target->getGPUForRead();
    FloatGPUMirroredMemoryBlock *pattern_errors = 
      new FloatGPUMirroredMemoryBlock(target->getSize());
    float *pattern_errors_ptr = pattern_errors->getGPUForWrite();
    dim3 block, grid;
    computeBlockAndGridSizesForAColumnMajorBunch(bunch_size, size,
						 block, grid);
    computeMAELossFunctionKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (input_ptr,
       target_ptr,
       pattern_errors_ptr,
       zero_epsilon_distance,
       bunch_size,
       bunch_size,
       size);
    sumBunchPatternErrors(loss_output->getGPUForWrite(),
			  pattern_errors_ptr,
			  bunch_size, size);
    delete pattern_errors;
  }
  else {
#endif

    COMPUTE_LOSS(input,target,loss_output,size,bunch_size,
		 float absd = fabsf(input_ptr[b] - target_ptr[b]);
		 if (absd < zero_epsilon_distance) absd = 0.0f,
		 absd);
    
#ifdef USE_CUDA
  }
#endif
}


void doComputeMAEGradient(FloatGPUMirroredMemoryBlock *input,
			  FloatGPUMirroredMemoryBlock *target,
			  FloatGPUMirroredMemoryBlock *error_output,
			  float zero_epsilon_distance,
			  unsigned int size,
			  unsigned int bunch_size,
			  bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {    
    const float *input_ptr  = input->getGPUForRead();
    const float *target_ptr = target->getGPUForRead();
    float *error_output_ptr = error_output->getGPUForWrite();
    dim3 block, grid;
    computeBlockAndGridSizesForAColumnMajorBunch(bunch_size, size,
						 block, grid);
    computeMAEGradientKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (input_ptr,
       target_ptr,
       error_output_ptr,
       zero_epsilon_distance,
       bunch_size,
       bunch_size,
       size,
       1.0f/size);
  }
  else {
#endif
    float d = 0.0f;
    const float *input_ptr  = input->getPPALForRead();
    const float *target_ptr = target->getPPALForRead();
    float *error_output_ptr = error_output->getPPALForWrite();
    float invN = 1.0f/size;
    for (unsigned int i = 0; i < size; i++) {
      for (unsigned int b=0; b<bunch_size; ++b) {
	d = input_ptr[b] - target_ptr[b];
	if (fabsf(d) < zero_epsilon_distance) error_output_ptr[b] = 0.0f;
	else {
	  if (d < 0.0f) error_output_ptr[b] = -invN;
	  else error_output_ptr[b] = invN;
	}
      }
      input_ptr  += bunch_size;
      target_ptr += bunch_size;
      error_output_ptr += bunch_size;
    }
#ifdef USE_CUDA
  }
#endif
}

void doCrossEntropyLossFunction(FloatGPUMirroredMemoryBlock *input,
				FloatGPUMirroredMemoryBlock *target,
				FloatGPUMirroredMemoryBlock *loss_output,
				float EPSILON,
				unsigned int size,
				unsigned int bunch_size,
				bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {    
    const float *input_ptr  = input->getGPUForRead();
    const float *target_ptr = target->getGPUForRead();
    FloatGPUMirroredMemoryBlock *pattern_errors = 
      new FloatGPUMirroredMemoryBlock(target->getSize());
    float *pattern_errors_ptr = pattern_errors->getGPUForWrite();
    dim3 block, grid;
    computeBlockAndGridSizesForAColumnMajorBunch(bunch_size, size,
						 block, grid);
    computeCrossEntropyLossFunctionKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (input_ptr,
       target_ptr,
       pattern_errors_ptr,
       EPSILON,
       bunch_size,
       bunch_size,
       size);
    sumBunchPatternErrors(loss_output->getGPUForWrite(),
			  pattern_errors_ptr,
			  bunch_size, size);
    delete pattern_errors;
  }
  else {
#endif
    float log_epsilon   = logf(EPSILON);
    float log_1_epsilon = logf(1.0f - EPSILON);

    COMPUTE_LOSS(input,target,loss_output,size,bunch_size,
		 april_assert(!(input_ptr[b] > 0.0f) &&
			      "Only log-based activation functions are allowed");
		 april_assert(!(target_ptr[b] < 0.0f) && !(target_ptr[b] > 1.0f) &&
			      "Only [0,1] target patterns are allowed");
		 // compute derivative
		 float  log_o     = clamp(input_ptr[b], log_epsilon, log_1_epsilon);
		 double o         = exp(input_ptr[b]);
		 float  log_inv_o = log(1.0 - o);
		 // CLAMP of reference (target)
		 float  t         = clamp(target_ptr[b], EPSILON, 1.0f - EPSILON);
		 // CLAMP of 1.0 - reference (target). We do clamp again to avoid
		 // numerical approximation problems, and to ensure correct working of
		 // inv_t > EPSILON comparison
		 float  inv_t     = clamp(1.0f - target_ptr[b], EPSILON, 1.0f - EPSILON);
		 // printf("%g * %g :: %g * %g :: %g\n", t, log_o, inv_t, log_inv_o, o);
		 float sum;
		 if (t > EPSILON) sum = -t * log_o;
		 else sum = 0.0f;
		 if (inv_t > EPSILON) sum -= inv_t * log_inv_o,
		 sum);
#ifdef USE_CUDA
  }
#endif
}

void doMultiClassCrossEntropyLossFunction(FloatGPUMirroredMemoryBlock *input,
					  FloatGPUMirroredMemoryBlock *target,
					  FloatGPUMirroredMemoryBlock *loss_output,
					  float EPSILON,
					  unsigned int size,
					  unsigned int bunch_size,
					  bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    const float *input_ptr  = input->getGPUForRead();
    const float *target_ptr = target->getGPUForRead();
    FloatGPUMirroredMemoryBlock *pattern_errors = 
      new FloatGPUMirroredMemoryBlock(target->getSize());
    float *pattern_errors_ptr = pattern_errors->getGPUForWrite();
    dim3 block, grid;
    computeBlockAndGridSizesForAColumnMajorBunch(bunch_size, size,
						 block, grid);
    computeMultiClassCrossEntropyLossFunctionKernel<<<grid, block, 0,
      GPUHelper::getCurrentStream()>>>
      (input_ptr,
       target_ptr,
       pattern_errors_ptr,
       EPSILON,
       bunch_size,
       bunch_size,
       size);
    sumBunchPatternErrors(loss_output->getGPUForWrite(),
			  pattern_errors_ptr,
			  bunch_size, size);
    delete pattern_errors;
  }
  else {
#endif
    COMPUTE_LOSS(input,target,loss_output,size,bunch_size,
		 april_assert(!(input_ptr[b] > 0.0f) &&
			      "Only log-based activation functions are allowed");
		 april_assert(!(target_ptr[b] < 0.0f) && !(target_ptr[b] > 1.0f) &&
			      "Only [0,1] target patterns are allowed");
		 // compute derivative
		 float log_o = input_ptr[b];
		 float t = clamp(target_ptr[b], EPSILON, 1.0f - EPSILON);
		 float sum;
		 if (t > EPSILON) sum = -t * log_o;
		 else sum = 0.0f,
		 sum);
#ifdef USE_CUDA
  }
#endif
}

void doComputeCrossEntropyGradient(FloatGPUMirroredMemoryBlock *input,
				   FloatGPUMirroredMemoryBlock *target,
				   FloatGPUMirroredMemoryBlock *error_output,
				   float EPSILON,
				   unsigned int size,
				   unsigned int bunch_size,
				   bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {    
    const float *input_ptr  = input->getGPUForRead();
    const float *target_ptr = target->getGPUForRead();
    float *error_output_ptr = error_output->getGPUForWrite();
    dim3 block, grid;
    computeBlockAndGridSizesForAColumnMajorBunch(bunch_size, size,
						 block, grid);
    computeCrossEntropyGradientKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (input_ptr,
       target_ptr,
       error_output_ptr,
       EPSILON,
       bunch_size,
       bunch_size,
       size);
  }
  else {
#endif
    UNUSED_VARIABLE(EPSILON);
    const float *input_ptr  = input->getPPALForRead();
    const float *target_ptr = target->getPPALForRead();
    float *error_output_ptr = error_output->getPPALForWrite();
    for (unsigned int i = 0; i < size; i++) {
      for (unsigned int b=0; b<bunch_size; ++b)
	error_output_ptr[b] = expf(input_ptr[b]) - target_ptr[b];
      input_ptr  += bunch_size;
      target_ptr += bunch_size;
      error_output_ptr += bunch_size;
    }
#ifdef USE_CUDA
  }
#endif
}

/*
  void doCalculateTanhErrorFunction(FloatGPUMirroredMemoryBlock *output,
  FloatGPUMirroredMemoryBlock *target_output,
  FloatGPUMirroredMemoryBlock *output_error,
  FloatGPUMirroredMemoryBlock *pattern_errors,
  unsigned int output_size,
  const ANNConfiguration &conf,
  bool use_gpu) {
  #ifdef USE_CUDA
  if (use_gpu) {
  const float *output_ptr        = output->getGPUForRead();
  const float *target_output_ptr = target_output->getGPUForRead();
  float *output_error_ptr        = output_error->getGPUForWrite();
  float *pattern_errors_ptr      = pattern_errors->getGPUForWrite();
  dim3 block, grid;
  computeBlockAndGridSizesForAColumnMajorBunch(conf, output_size,
  block, grid);
  
  applyTanhErrorFunctionKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
  (output_ptr,
  target_output_ptr,
  output_error_ptr,
  pattern_errors_ptr,
  conf.cur_bunch_size,
  conf.max_bunch_size,
  output_size);
  }
  else {
  #endif
  float d = 0;
  const float *output_ptr        = output->getPPALForRead();
  const float *target_output_ptr = target_output->getPPALForRead();
  float *output_error_ptr        = output_error->getPPALForWrite();
  float *pattern_errors_ptr      = pattern_errors->getPPALForWrite();
    
  for (unsigned int i = 0; i < output_size; i++) {
  for (unsigned int b=0; b<conf.cur_bunch_size; ++b) {
  d = output_error_ptr[b] = output_ptr[b] - target_output_ptr[b];
  if (d < -0.9999999f)
  output_error_ptr[b] = -DERIVATIVE_SATURATION;
  else if (d > 0.9999999f)
  output_error_ptr[b] =  DERIVATIVE_SATURATION;
  else output_error_ptr[b] = log((1.0f+output_error_ptr[b])/(1.0f-output_error_ptr[b]));
  pattern_errors_ptr[b] += d*d;
  }
  output_ptr         += conf.max_bunch_size;
  target_output_ptr  += conf.max_bunch_size;
  output_error_ptr   += conf.max_bunch_size;
  pattern_errors_ptr += conf.max_bunch_size;
  }
  #ifdef USE_CUDA
  }
  #endif
  }
*/

/*
  void doCalculateMixtureCrossEntropy(FloatGPUMirroredMemoryBlock *output,
  FloatGPUMirroredMemoryBlock *target_output,
  FloatGPUMirroredMemoryBlock *output_error,
  FloatGPUMirroredMemoryBlock *pattern_errors,
  float EPSILON,
  float INF,
  unsigned int output_size,
  const ANNConfiguration &conf,
  bool use_gpu) {
  const float *output_ptr        = output->getPPALForRead();
  const float *target_output_ptr = target_output->getPPALForRead();
  float *output_error_ptr        = output_error->getPPALForWrite();
  float *pattern_errors_ptr      = pattern_errors->getGPUForWrite();

  for (unsigned int b=0; b<conf.cur_bunch_size; ++b) {
  float Z = 0.0f;
  unsigned int ipos = b;
  for (unsigned int i=0; i<output_size; ++i)
  {
  Z += target_output_ptr[ipos] * output_ptr[ipos];
  ipos += conf.max_bunch_size;
  }
  Z = 1.0f/Z;
  float prob = 0.0f;
  ipos = b;
  for (unsigned int i = 0; i < output_size; i++) {
  float component_prob = target_output_ptr[ipos] * output_ptr[ipos];
  output_error_ptr[ipos] = output_ptr[ipos] - component_prob*Z;
  prob += component_prob;
  ipos += conf.max_bunch_size;
  }
  s += ((fabs(prob) > EPSILON) ? logf(prob) : INF);
  }
  return s;
  }
*/

// F(o,t) = (1 + beta^2) * sum o_i * t_i / sum( o_i + beta^2 * t_i )
// Gab = (1 + beta^2) sum o_i * t_i
// Hab = sum( o_i + beta^2 * t_i )
void doLocalFMeasureLossFunction(FloatGPUMirroredMemoryBlock *input,
				 FloatGPUMirroredMemoryBlock *target,
				 FloatGPUMirroredMemoryBlock *loss_output,
				 unsigned int size,
				 unsigned int bunch_size,
				 float beta,
				 float &Gab, float &Hab,
				 bool complement_output,
				 bool use_gpu) {
  if (use_gpu)   ERROR_EXIT(128, "GPU VERSION NOT IMPLEMENTED YET!!!\n");
  if (size != 1) ERROR_EXIT(128, "Multi-class version is not implemented\n");
  const float *input_ptr  = input->getPPALForRead();
  const float *target_ptr = target->getPPALForRead();
  float *loss_output_ptr  = loss_output->getPPALForWrite();
  Gab = 0.0f;
  Hab = 0.0f;
  float beta2 = beta*beta;
  for (unsigned int b=0; b<bunch_size; ++b) {
    unsigned int ipos = b;
    for (unsigned int i = 0; i < size; i++) {
      // float out = clamp(output_ptr[ipos], 0.0f, 1.0f);
      float in = input_ptr[ipos];
      assert(!(in < 0.0f) && !(in > 1.0f) &&
	     "Only [0,1] activation functions are allowed");
      if (!complement_output) {
	Gab += in * target_ptr[ipos];
	Hab += in + beta2 * target_ptr[ipos];
      }
      else {
	Gab += 1.0f + in * target_ptr[ipos] - in - target_ptr[ipos];
	Hab += 1.0f + beta2 - in - beta2 * target_ptr[ipos];
      }
      ipos += bunch_size;
    }
  }
  Gab = (1.0f + beta2)*Gab;
  // cambiamos de signo para convertir la minimizacion en una maximizacion
  float error;
  if (Hab > 0.0f)
    error = -Gab/Hab;
  else error = -1.0f;
  *loss_output_ptr = error;
}

// F'(o,t) = (1 + beta^2)*t_i / Hab - Gab/Hab
void doComputeLocalFMeasureGradient(FloatGPUMirroredMemoryBlock *target,
				    FloatGPUMirroredMemoryBlock *output_error,
				    unsigned int size,
				    unsigned int bunch_size,
				    float beta,
				    float Gab, float Hab,
				    bool complement_output,
				    bool use_gpu) {
  if (use_gpu)   ERROR_EXIT(128, "GPU VERSION NOT IMPLEMENTED!!!\n");
  if (size != 1) ERROR_EXIT(128, "Multi-class version is not implemented\n");
  const float *target_ptr = target->getPPALForRead();
  float *output_error_ptr = output_error->getPPALForWrite();
  float beta2_p1  = 1.0f + beta*beta;
  if (Hab > 0.0f) {
    float inv_Hab     = 1.0f/Hab;
    float Gab_DIV_Hab2 = Gab*inv_Hab*inv_Hab;
    for (unsigned int b=0; b<bunch_size; ++b) {
      unsigned int ipos = b;
      for (unsigned int i = 0; i < size; i++) {
	float t = target_ptr[ipos];
	if (complement_output) t = 1.0f - t;
	output_error_ptr[ipos] = beta2_p1*t*inv_Hab - Gab_DIV_Hab2;
	ipos += bunch_size;
      }
    }
  }
}

/*
  void doCalculateGA(FloatGPUMirroredMemoryBlock *output,
  FloatGPUMirroredMemoryBlock *target_output,
  FloatGPUMirroredMemoryBlock *output_error,
  FloatGPUMirroredMemoryBlock *pattern_errors,
  unsigned int output_size,
  const ANNConfiguration &conf,
  bool use_gpu) {
  const float *output_ptr        = output->getPPALForRead();
  const float *target_output_ptr = target_output->getPPALForRead();
  float *output_error_ptr        = output_error->getPPALForWrite();

  for (unsigned int b=0; b<conf.cur_bunch_size; ++b) {
  // Las 2 siguientes variables no se emplean?
  //float sum_a_b = 0.0f;
  //float sum_c_a_b;
  float Gab = 0.0f, Hab = 0.0f;
  unsigned int ipos = b;
  for (unsigned int i = 0; i < output_size; i++) {
  Gab += output_ptr[ipos] * target_output_ptr[ipos];
  Hab += output_ptr[ipos] + target_output_ptr[ipos];
  ipos += conf.max_bunch_size;
  }
  Gab *= 2.0f;
  s   += 1.0f - Gab/Hab; // hacemos 1 - FMeasure para cambiar la minimizacion
  // por una maximizacion
  float HabP2 = Hab*Hab;
  ipos = b;
  for (unsigned int i = 0; i < output_size; i++) {
  // Aqui cambiamos de signo para convertir una minimizacion en una
  // maximizacion
  output_error_ptr[ipos] = -(2 * target_output_ptr[ipos] * Hab - Gab) / HabP2;
  ipos += conf.max_bunch_size;
  }
  }
  return s;
  }

*/

#undef sigmoid
#undef clip
