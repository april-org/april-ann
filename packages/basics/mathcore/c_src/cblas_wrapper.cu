#include "hip/hip_runtime.h"
/*
 * This file is part of the Neural Network modules of the APRIL toolkit (A
 * Pattern Recognizer In Lua).
 *
 * Copyright 2012, Salvador España-Boquera, Adrian Palacios Corella, Francisco
 * Zamora-Martinez
 *
 * The APRIL-ANN toolkit is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License version 3 as
 * published by the Free Software Foundation
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License
 * for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this library; if not, write to the Free Software Foundation,
 * Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA
 *
 */
#include <cmath>
#include "unused_variable.h"
#include "wrapper.h"
#include "cuda_utils.h"

#define fsignf(x) ((x)<0.0f) ? (-1.0f) : ( ((x)>0.0f) ? (1.0f) : (0.0f) )

///////////////////////////////////////////////////////////
/////////////////// Kernels ///////////////////////////////
///////////////////////////////////////////////////////////

#ifdef USE_CUDA

#define CWISE_FUNC_KERNEL(func) template<typename T>		\
  __global__ void						\
  func##FuncKernel(T *v, unsigned int N, unsigned int stride) {	\
    unsigned int x_idx = blockIdx.x * blockDim.x + threadIdx.x;	\
    if (x_idx < N) {						\
      T *aux = v + x_idx*stride;				\
      *aux = func(*aux);					\
    }								\
  }

CWISE_FUNC_KERNEL(logf);
CWISE_FUNC_KERNEL(log1pf);
CWISE_FUNC_KERNEL(expf);
CWISE_FUNC_KERNEL(sqrtf);
CWISE_FUNC_KERNEL(tanf);
CWISE_FUNC_KERNEL(tanhf);
CWISE_FUNC_KERNEL(atanf);
CWISE_FUNC_KERNEL(atanhf);
CWISE_FUNC_KERNEL(sinf);
CWISE_FUNC_KERNEL(sinhf);
CWISE_FUNC_KERNEL(asinf);
CWISE_FUNC_KERNEL(asinhf);
CWISE_FUNC_KERNEL(cosf);
CWISE_FUNC_KERNEL(coshf);
CWISE_FUNC_KERNEL(acosf);
CWISE_FUNC_KERNEL(acoshf);
CWISE_FUNC_KERNEL(fabsf);
CWISE_FUNC_KERNEL(fsignf);

#undef CWISE_FUNC_KERNEL

template<typename T>
__global__ void complementFuncKernel(T *v, unsigned int N, unsigned int stride) {
  unsigned int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (x_idx < N) {
    T *aux = v + x_idx*stride;
    *aux = (1.0f - *aux);
  }
}

#define CWISE_FUNC_KERNEL(func) template<typename T>		\
  __global__ void						\
  func##FuncKernel(T *v, unsigned int N, unsigned int stride,	\
		   T value) {					\
    unsigned int x_idx = blockIdx.x * blockDim.x + threadIdx.x;	\
    if (x_idx < N) {						\
      T *aux = v + x_idx*stride;				\
      *aux = func(*aux, value);					\
    }								\
  }

CWISE_FUNC_KERNEL(powf);

#undef CWISE_FUNC_KERNEL

#endif

///////////////////////////////////////////////////////////
//////////////////// BLAS wrappers ////////////////////////
///////////////////////////////////////////////////////////

void doPLogP(unsigned int N,
	     FloatGPUMirroredMemoryBlock *v,
	     unsigned int stride,
	     unsigned int shift,
	     bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    ERROR_PRINT("CUDA VERSION NOT IMPLEMENTED\n");
  }
  //  else {
#endif
  float *v_mem = v->getPPALForReadAndWrite() + shift;
  for (unsigned int i=0; i<N; ++i, v_mem += stride)
    if (*v_mem > 0.0f || *v_mem < 0.0f) *v_mem = (*v_mem) * logf(*v_mem);
#ifdef USE_CUDA
  //  }
#endif  
}

void doLog(unsigned int N,
	   FloatGPUMirroredMemoryBlock *v,
	   unsigned int stride,
	   unsigned int shift,
	   bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    logfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = logf(*v_mem);
#ifdef USE_CUDA
  }
#endif  
}

void doLog1p(unsigned int N,
	     FloatGPUMirroredMemoryBlock *v,
	     unsigned int stride,
	     unsigned int shift,
	     bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    log1pfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = log1pf(*v_mem);
#ifdef USE_CUDA
  }
#endif  
}

void doExp(unsigned int N,
	   FloatGPUMirroredMemoryBlock *v,
	   unsigned int stride,
	   unsigned int shift,
	   bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    expfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = expf(*v_mem);
#ifdef USE_CUDA
  }
#endif  
}

void doSqrt(unsigned int N,
	    FloatGPUMirroredMemoryBlock *v,
	    unsigned int stride,
	    unsigned int shift,
	    bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    sqrtfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = sqrtf(*v_mem);
#ifdef USE_CUDA
  }
#endif  
}

void doTan(unsigned int N,
	   FloatGPUMirroredMemoryBlock *v,
	   unsigned int stride,
	   unsigned int shift,
	   bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    tanfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = tanf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doTanh(unsigned int N,
	    FloatGPUMirroredMemoryBlock *v,
	    unsigned int stride,
	    unsigned int shift,
	    bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    tanhfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = tanhf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doAtan(unsigned int N,
	    FloatGPUMirroredMemoryBlock *v,
	    unsigned int stride,
	    unsigned int shift,
	    bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    atanfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = atanf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doAtanh(unsigned int N,
	     FloatGPUMirroredMemoryBlock *v,
	     unsigned int stride,
	     unsigned int shift,
	     bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    atanhfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = atanhf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doSin(unsigned int N,
	   FloatGPUMirroredMemoryBlock *v,
	   unsigned int stride,
	   unsigned int shift,
	   bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    sinfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = sinf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doSinh(unsigned int N,
	    FloatGPUMirroredMemoryBlock *v,
	    unsigned int stride,
	    unsigned int shift,
	    bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    sinhfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = sinhf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doAsin(unsigned int N,
	    FloatGPUMirroredMemoryBlock *v,
	    unsigned int stride,
	    unsigned int shift,
	    bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    asinfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = asinf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doAsinh(unsigned int N,
	     FloatGPUMirroredMemoryBlock *v,
	     unsigned int stride,
	     unsigned int shift,
	     bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    asinhfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = asinhf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doCos(unsigned int N,
	   FloatGPUMirroredMemoryBlock *v,
	   unsigned int stride,
	   unsigned int shift,
	   bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    cosfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = cosf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doCosh(unsigned int N,
	    FloatGPUMirroredMemoryBlock *v,
	    unsigned int stride,
	    unsigned int shift,
	    bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    coshfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = coshf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doAcos(unsigned int N,
	    FloatGPUMirroredMemoryBlock *v,
	    unsigned int stride,
	    unsigned int shift,
	    bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    acosfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = acosf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doAcosh(unsigned int N,
	     FloatGPUMirroredMemoryBlock *v,
	     unsigned int stride,
	     unsigned int shift,
	     bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    acoshfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = acoshf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doAbs(unsigned int N,
	   FloatGPUMirroredMemoryBlock *v,
	   unsigned int stride,
	   unsigned int shift,
	   bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    fabsfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = fabsf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doComplement(unsigned int N,
		  FloatGPUMirroredMemoryBlock *v,
		  unsigned int stride,
		  unsigned int shift,
		  bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    complementFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = (1.0f - *v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doSign(unsigned int N,
	    FloatGPUMirroredMemoryBlock *v,
	    unsigned int stride,
	    unsigned int shift,
	    bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    fsignfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = fsignf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}


void doPow(unsigned int N,
	   FloatGPUMirroredMemoryBlock *v,
	   unsigned int stride,
	   unsigned int shift,
	   float value,
	   bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    powfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride, value);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride)
      *v_mem = powf(*v_mem, value);
#ifdef USE_CUDA
  }
#endif
}
