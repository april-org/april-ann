#include "hip/hip_runtime.h"
/*
 * This file is part of the Neural Network modules of the APRIL toolkit (A
 * Pattern Recognizer In Lua).
 *
 * Copyright 2012, Salvador España-Boquera, Adrian Palacios Corella, Francisco
 * Zamora-Martinez
 *
 * The APRIL-MLP toolkit is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License version 3 as
 * published by the Free Software Foundation
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License
 * for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this library; if not, write to the Free Software Foundation,
 * Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA
 *
 */
#include "wrapper.h"
#include "cuda_utils.h"
#include "clamp.h"
#include "unused_variable.h"

#ifdef USE_CUDA
/***************************************
 ************** CUDA SECTION ***********
 ***************************************/

template<typename T>
__global__ void clampKernel(T *v, unsigned int N, unsigned int stride,
			    T lower, T upper) {
  unsigned int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (x_idx < N) {
    unsigned int x_pos = x_idx*stride;
    if (v[x_pos] < lower) v[x_pos] = lower;
    else if (v[x_pos] > upper) v[x_pos] = upper;
  }
}
#endif

/***************************************
 *********** TEMPLATE SECTION **********
 ***************************************/

template <typename T>
void doClamp(unsigned int N,
	     GPUMirroredMemoryBlock<T> *v,
	     unsigned int stride,
	     unsigned int shift,
	     T lower, T upper,
	     bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    T *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    clampKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride, lower, upper);
  }
  else {
#endif
    T *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride)
      *v_mem = april_utils::clamp(*v_mem,lower,upper);
#ifdef USE_CUDA
  }
#endif
}

template void doClamp<float>(unsigned int N,
			     GPUMirroredMemoryBlock<float> *v,
			     unsigned int stride,
			     unsigned int shift,
			     float lower, float upper,
			     bool use_gpu);
