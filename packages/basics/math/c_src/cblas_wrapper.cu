#include "hip/hip_runtime.h"
/*
 * This file is part of the Neural Network modules of the APRIL toolkit (A
 * Pattern Recognizer In Lua).
 *
 * Copyright 2012, Salvador España-Boquera, Adrian Palacios Corella, Francisco
 * Zamora-Martinez
 *
 * The APRIL-MLP toolkit is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License version 3 as
 * published by the Free Software Foundation
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License
 * for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this library; if not, write to the Free Software Foundation,
 * Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA
 *
 */
#include <cmath>
#include "wrapper.h"
#include "clamp.h"
#include "ceiling_power_of_two.h"
#include "cuda_utils.h"

using april_utils::ceilingPowerOfTwo;

///////////////////////////////////////////////////////////
/////////////////// Kernels ///////////////////////////////
///////////////////////////////////////////////////////////

#ifdef USE_CUDA
__global__ void sumVectorFirstReduction(const float *v,
					float *sums,
					unsigned int reduction_top,
					unsigned int size,
					unsigned int stride) {
  unsigned int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int active_reduction = reduction_top >> 1;
  if (x_idx < size && x_idx < active_reduction) {
    unsigned int x_pos = x_idx * stride;
    unsigned int passive_index = (x_idx + active_reduction) * stride;
    if (x_idx + active_reduction < size)
      sums[x_pos] = v[x_pos] + v[passive_index];
    else
      sums[x_pos] = v[x_pos];
  }
}

__global__ void sumVectorNextReduction(float *sums,
				       unsigned int reduction_top,
				       unsigned int size,
				       unsigned int stride) {
  unsigned int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int active_reduction = reduction_top >> 1;

  if (x_idx < size && x_idx < active_reduction) {
    unsigned int index = x_idx*stride;
    unsigned int passive_index = (x_idx+active_reduction)*stride;
    sums[index] = sums[index] + sums[passive_index];
  }
}

__global__ void clampKernel(float *v, unsigned int N, unsigned int stride,
			    float lower, float upper) {
  unsigned int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (x_idx < N) {
    unsigned int x_pos = x_idx*stride;
    if (v[x_pos] < lower) v[x_pos] = lower;
    else if (v[x_pos] > upper) v[x_pos] = upper;
  }
}

__global__ void scalarAddKernel(float *v, unsigned int N, unsigned int stride,
				float value) {
  unsigned int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (x_idx < N) {
    float *aux = v + x_idx*stride;
    *aux += value;
  }
}

__global__ void fillKernel(float *v, unsigned int N, unsigned int stride,
			   float value) {
  unsigned int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (x_idx < N) v[x_idx*stride] = value;
}

#define CWISE_FUNC_KERNEL(func) __global__ void				\
  func##FuncKernel(float *v, unsigned int N, unsigned int stride) {	\
    unsigned int x_idx = blockIdx.x * blockDim.x + threadIdx.x;		\
    if (x_idx < N) {							\
      float *aux = v + x_idx*stride;					\
      *aux = func(*aux);						\
    }									\
  }

CWISE_FUNC_KERNEL(logf);
CWISE_FUNC_KERNEL(log1pf);
CWISE_FUNC_KERNEL(expf);
CWISE_FUNC_KERNEL(sqrtf);
CWISE_FUNC_KERNEL(tanhf);

#undef CWISE_FUNC_KERNEL

#define CWISE_FUNC_KERNEL(func) __global__ void				\
  func##FuncKernel(float *v, unsigned int N, unsigned int stride,	\
		   float value) {					\
    unsigned int x_idx = blockIdx.x * blockDim.x + threadIdx.x;		\
    if (x_idx < N) {							\
      float *aux = v + x_idx*stride;					\
      *aux = func(*aux, value);						\
    }									\
  }

CWISE_FUNC_KERNEL(powf);

#undef CWISE_FUNC_KERNEL

__global__ void scopyLoopKernel(unsigned int N,
				const float *x_mem,
				unsigned int x_inc,
				float *y_mem,
				unsigned int y_inc,
				unsigned int times,
				unsigned int y_ld) {
  
  unsigned int matrix_x_pos, matrix_y_pos;
  matrix_x_pos = blockIdx.x*blockDim.x + threadIdx.x;
  matrix_y_pos = blockIdx.y*blockDim.y + threadIdx.y;
  if (matrix_x_pos < times && matrix_y_pos < N) {
    unsigned int index_x = matrix_y_pos*x_inc;
    unsigned int index_y = matrix_x_pos*y_ld + matrix_y_pos*y_inc;
    y_mem[index_y] = x_mem[index_x];
  }
}

__global__ void saxpyLoopKernel(unsigned int N,
				float alpha,
				const float *x_mem,
				unsigned int x_inc,
				float *y_mem,
				unsigned int y_inc,
				unsigned int times,
				unsigned int x_ld,
				unsigned int y_ld) {
  unsigned int matrix_x_pos, matrix_y_pos;
  matrix_x_pos = blockIdx.x*blockDim.x + threadIdx.x;
  matrix_y_pos = blockIdx.y*blockDim.y + threadIdx.y;
  if (matrix_x_pos < times && matrix_y_pos < N) {
    unsigned int index_x = matrix_x_pos*x_ld + matrix_y_pos*x_inc;
    unsigned int index_y = matrix_x_pos*y_ld + matrix_y_pos*y_inc;
    float val = alpha * x_mem[index_x];
    // This loop is used to synchronize the threads for accessing
    // the global memory where they write the results. The loop
    // gets all the values from the threads at the index X in 
    // the current block, synchronizing the access to Y.
    for (unsigned int i=0; i<blockDim.x; ++i) {
      if (i==threadIdx.x) y_mem[index_y] += val;
      __syncthreads();
    }
  }
}
#endif

///////////////////////////////////////////////////////////
//////////////////// BLAS wrappers ////////////////////////
///////////////////////////////////////////////////////////

void doSgemv(CBLAS_ORDER major_type, CBLAS_TRANSPOSE a_transpose,
	     int m, int n,
	     float alpha, FloatGPUMirroredMemoryBlock *a, unsigned int a_inc,
	     FloatGPUMirroredMemoryBlock *x, unsigned int x_inc,
	     float beta, FloatGPUMirroredMemoryBlock *y, unsigned int y_inc,
	     unsigned int a_shift, unsigned int x_shift, unsigned int y_shift,
	     bool use_gpu) {
  const float *a_mem, *x_mem;
  float *y_mem;
#ifdef USE_CUDA
  if (use_gpu) {
    hipblasStatus_t status;
    hipblasHandle_t handle = GPUHelper::getHandler();
    assert(major_type == CblasColMajor);
    hipblasOperation_t cublas_a_transpose = getCublasOperation(a_transpose);
    a_mem = a->getGPUForRead() + a_shift;
    x_mem = x->getGPUForRead() + x_shift;
    y_mem = y->getGPUForReadAndWrite() + y_shift;

    status = hipblasSetStream(handle, GPUHelper::getCurrentStream());
    checkCublasError(status);

    status = hipblasSgemv(handle, cublas_a_transpose,
			 m, n,
			 &alpha, a_mem, a_inc,
			 x_mem, x_inc,
			 &beta, y_mem, y_inc);

    checkCublasError(status);
  }
  else {
#endif
    a_mem = a->getPPALForRead() + a_shift;
    x_mem = x->getPPALForRead() + x_shift;
    y_mem = y->getPPALForReadAndWrite() + y_shift;
    cblas_sgemv(major_type, a_transpose,
                m, n,
                alpha, a_mem, a_inc,
                x_mem, x_inc,
                beta, y_mem, y_inc);
#ifdef USE_CUDA
  }
#endif
}


void doScopy(int N, const FloatGPUMirroredMemoryBlock* x,
	     unsigned int x_shift,
	     unsigned int x_inc,
	     FloatGPUMirroredMemoryBlock* y,
	     unsigned int y_shift,
	     unsigned int y_inc,
	     bool use_gpu)
{
  const float *x_mem;
  float *y_mem;
#ifdef USE_CUDA
  if (use_gpu) {
    hipblasStatus_t status;
    hipblasHandle_t handle = GPUHelper::getHandler();
    //printf("Doing a scopy with comp=1 & cuda=1\n");
    x_mem = x->getGPUForRead() + x_shift;
    y_mem = y->getGPUForWrite() + y_shift;
    
    status = hipblasSetStream(handle, GPUHelper::getCurrentStream());
    checkCublasError(status);
    
    status = hipblasScopy(handle, N, x_mem, x_inc, y_mem, y_inc);
    
    checkCublasError(status);
  }
  else {
    //printf("Doing a scopy with comp=1 & cuda=0\n");
#endif
#ifndef USE_CUDA
    //printf("Doing a scopy with comp=0 & cuda=0\n");
#endif
    x_mem = x->getPPALForRead() + x_shift;
    y_mem = y->getPPALForWrite() + y_shift;

    cblas_scopy(N, x_mem, x_inc, y_mem, y_inc);
#ifdef USE_CUDA
  }
#endif
}

void doScopyLoop(int N,
		 FloatGPUMirroredMemoryBlock* x,
		 unsigned int x_inc,
		 FloatGPUMirroredMemoryBlock* y,
		 unsigned int y_inc,
		 unsigned int times,
		 const unsigned int stride,
		 bool use_gpu)
{
  const float *x_mem;
  float *y_mem;
#ifdef USE_CUDA
  if (use_gpu) {
    //printf("Doing a scopy with comp=1 & cuda=1\n");
    x_mem = x->getGPUForRead();
    y_mem = y->getGPUForWrite();

    const unsigned int MAX_THREADS = GPUHelper::getMaxThreadsPerBlock();
    dim3 block, grid;
    // Number of threads on each block dimension
    block.x = min(MAX_THREADS, times);
    block.y = min(MAX_THREADS/block.x, N);
    block.z = 1;

    grid.x = (times/block.x +
	      (times % block.x ? 1 : 0));
    grid.y = (N/block.y + (N % block.y ? 1 : 0));
    grid.z = 1;

    scopyLoopKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (N, x_mem, x_inc, y_mem, y_inc, times, stride);
  }
  else {
    //printf("Doing a scopy with comp=1 & cuda=0\n");
#endif
#ifndef USE_CUDA
    //printf("Doing a scopy with comp=0 & cuda=0\n");
#endif
    x_mem = x->getPPALForRead();
    y_mem = y->getPPALForWrite();

    for (unsigned int i = 0; i < times; i++)
      cblas_scopy(N, 
		  x_mem, x_inc,
		  y_mem + i * stride , y_inc);
#ifdef USE_CUDA
  }
#endif
}

void doSaxpy(int N,
	     float alpha,
	     const FloatGPUMirroredMemoryBlock* x,
	     unsigned int x_shift,
	     unsigned int x_inc,
	     FloatGPUMirroredMemoryBlock* y,
	     unsigned int y_shift,
	     unsigned int y_inc,
	     bool use_gpu)
{
  const float *x_mem;
  float *y_mem;
#ifdef USE_CUDA
  if (use_gpu) {
    hipblasStatus_t status;
    hipblasHandle_t handle = GPUHelper::getHandler();
    //printf("Doing a saxpy with comp=1 & cuda=1\n");
    x_mem = x->getGPUForRead() + x_shift;
    y_mem = y->getGPUForReadAndWrite() + y_shift;

    status = hipblasSetStream(handle, GPUHelper::getCurrentStream());
    checkCublasError(status);

    status = hipblasSaxpy(handle, N, &alpha, x_mem, x_inc, y_mem, y_inc);

    checkCublasError(status);
  }
  else {
    //printf("Doing a saxpy with comp=1 & cuda=0\n");
#endif
#ifndef USE_CUDA
    //printf("Doing a saxpy with comp=0 & cuda=0\n");
#endif
    x_mem = x->getPPALForRead() + x_shift;
    y_mem = y->getPPALForReadAndWrite() + y_shift;

    cblas_saxpy(N, alpha, x_mem, x_inc, y_mem, y_inc);
#ifdef USE_CUDA
  }
#endif
}

void doSaxpyLoop(int N,
		 float alpha,
		 FloatGPUMirroredMemoryBlock* x,
		 unsigned int x_inc,
		 unsigned int x_shift,
		 FloatGPUMirroredMemoryBlock* y,
		 unsigned int y_inc,
		 unsigned int y_shift,
		 unsigned int times,
		 const unsigned int x_stride,
		 const unsigned int y_stride,
		 bool use_gpu)
{
  const float *x_mem;
  float *y_mem;
#ifdef USE_CUDA
  if (use_gpu) {
    x_mem = x->getGPUForRead() + x_shift;
    y_mem = y->getGPUForReadAndWrite() + y_shift;

    const unsigned int MAX_THREADS = GPUHelper::getMaxThreadsPerBlock();
    dim3 block, grid;
    // Number of threads on each block dimension
    block.x = min(MAX_THREADS, times);
    block.y = min(MAX_THREADS/block.x, N);
    block.z = 1;

    grid.x = (times/block.x +
	      (times % block.x ? 1 : 0));
    grid.y = (N/block.y + (N % block.y ? 1 : 0));
    grid.z = 1;

    saxpyLoopKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (N, alpha, x_mem, x_inc, y_mem, y_inc, times, x_stride, y_stride);
  }
  else {
    //printf("Doing a saxpy loop with comp=1 & cuda=0\n");
#endif
#ifndef USE_CUDA
    //printf("Doing a saxpy loop with comp=0 & cuda=0\n");
#endif
    x_mem = x->getPPALForRead() + x_shift;
    y_mem = y->getPPALForReadAndWrite() + y_shift;

    for (unsigned int i = 0; i < times; i++)
      cblas_saxpy(N, alpha,
		  x_mem + i * x_stride, x_inc, 
		  y_mem + i * y_stride, y_inc);
#ifdef USE_CUDA
  }
#endif
}

void doSgemm(CBLAS_ORDER major_type,
	     CBLAS_TRANSPOSE a_transpose,
	     CBLAS_TRANSPOSE b_transpose,
	     int m,
	     int n,
	     int k,
	     float alpha,
	     FloatGPUMirroredMemoryBlock* a,
	     unsigned int a_inc,
	     FloatGPUMirroredMemoryBlock* b,
	     unsigned int b_inc,
	     float beta,
	     FloatGPUMirroredMemoryBlock* c,
	     unsigned int c_inc,
	     unsigned int a_shift,
	     unsigned int b_shift,
	     unsigned int c_shift,
	     bool use_gpu)
{
  const float *a_mem, *b_mem;
  float *c_mem;
#ifdef USE_CUDA
  if (use_gpu) {
    hipblasStatus_t status;
    hipblasHandle_t handle = GPUHelper::getHandler();
    assert(major_type == CblasColMajor);
    //printf("Doing a sgemm with comp=1 & cuda=1\n");
    a_mem = a->getGPUForRead() + a_shift;
    b_mem = b->getGPUForRead() + b_shift;
    c_mem = c->getGPUForReadAndWrite() + c_shift;
    hipblasOperation_t cublas_a_transpose = getCublasOperation(a_transpose);
    hipblasOperation_t cublas_b_transpose = getCublasOperation(b_transpose);

    status = hipblasSetStream(handle, GPUHelper::getCurrentStream());
    checkCublasError(status);

    status = hipblasSgemm(handle, cublas_a_transpose, cublas_b_transpose,
			 m, n, k,
			 &alpha, a_mem, a_inc,
			 b_mem, b_inc,
			 &beta, c_mem, c_inc);

    checkCublasError(status);
  }
  else {
    //printf("Doing a sgemm with comp=1 & cuda=0\n");
#endif
    //printf("Doing a sgemm with comp=0 & cuda=0\n");
    a_mem = a->getPPALForRead() + a_shift;
    b_mem = b->getPPALForRead() + b_shift;
    c_mem = c->getPPALForReadAndWrite() + c_shift;

    // matrix matrix product: C = \alpha op(A) op(B) + \beta C
    cblas_sgemm(major_type,   // Row or Col Major
		a_transpose,  // Transpose or not A
		b_transpose,  // Transpose or not B
		m,            // num rows of A (before transpose)
		n,            // num rows at B (before transpose)
		k,            // Common dimension between A and B
		alpha,        // Alpha value
		a_mem,        // A matrix
		a_inc,        // A matrix stride
		b_mem,        // B matrix
		b_inc,        // B matrix stride
		beta,         // Beta value
		c_mem,        // C matrix
		c_inc);       // C matrix stride
#ifdef USE_CUDA
  }
#endif
}

void doSscal(unsigned int size,
	     FloatGPUMirroredMemoryBlock *x,
	     unsigned int inc,
	     unsigned int shift,
	     float alpha,
	     bool use_gpu) {
  float *x_mem;
#ifdef USE_CUDA
  if (use_gpu) {
    hipblasStatus_t status;
    hipblasHandle_t handle = GPUHelper::getHandler();
    x_mem = x->getGPUForReadAndWrite() + shift;

    status = hipblasSetStream(handle, GPUHelper::getCurrentStream());
    checkCublasError(status);

    status = hipblasSscal(handle, size, &alpha, x_mem, inc);

    checkCublasError(status);
  }
  else {
#endif
    x_mem = x->getPPALForReadAndWrite() + shift;
    cblas_sscal(size, alpha, x_mem, inc);
#ifdef USE_CUDA
  }
#endif
}

void doSger(CBLAS_ORDER major_type,
	    unsigned int m,
	    unsigned int n,
	    float alpha,
	    FloatGPUMirroredMemoryBlock *x,
	    unsigned int x_shift,
	    unsigned int x_inc,
	    FloatGPUMirroredMemoryBlock *y,
	    unsigned int y_shift,
	    unsigned int y_inc,
	    FloatGPUMirroredMemoryBlock *a,
	    unsigned int a_shift,
	    unsigned int a_inc,
	    bool use_gpu) {
  const float *x_mem;
  const float *y_mem;
  float *a_mem;
#ifdef USE_CUDA
  if (use_gpu) {
    hipblasStatus_t status;
    hipblasHandle_t handle = GPUHelper::getHandler();
    assert(major_type == CblasColMajor);
    x_mem = x->getGPUForRead() + x_shift;
    y_mem = y->getGPUForRead() + y_shift;
    a_mem = a->getGPUForReadAndWrite() + a_shift;

    status = hipblasSetStream(handle, GPUHelper::getCurrentStream());
    checkCublasError(status);

    status = hipblasSger(handle,
			m, n,
			&alpha,
			x_mem, x_inc,
			y_mem, y_inc,
			a_mem, a_inc);

    checkCublasError(status);
  }
  else {
#endif
    x_mem = x->getPPALForRead() + x_shift;
    y_mem = y->getPPALForRead() + y_shift;
    a_mem = a->getPPALForReadAndWrite() + a_shift;

    cblas_sger(major_type,
	       m, n,
	       alpha,
	       x_mem, x_inc,
	       y_mem, y_inc,
	       a_mem, a_inc);
#ifdef USE_CUDA
  }
#endif
}

float doSdot(unsigned int size,
	     const FloatGPUMirroredMemoryBlock *x,
	     unsigned int x_shift,
	     unsigned int x_inc,
	     const FloatGPUMirroredMemoryBlock *y,
	     unsigned int y_shift,
	     unsigned int y_inc,
	     bool use_gpu) {
  const float *x_mem;
  const float *y_mem;
  float ret;
#ifdef USE_CUDA
  if (use_gpu) {
    hipblasStatus_t status;
    hipblasHandle_t handle = GPUHelper::getHandler();
    x_mem = x->getGPUForRead() + x_shift;
    y_mem = y->getGPUForRead() + y_shift;
    
    status = hipblasSetStream(handle, GPUHelper::getCurrentStream());
    checkCublasError(status);
    
    status = hipblasSdot(handle,
			size,
			x_mem, x_inc,
			y_mem, y_inc,
			&ret);
    
    checkCublasError(status);
  }
  else {
#endif
    x_mem = x->getPPALForRead() + x_shift;
    y_mem = y->getPPALForRead() + y_shift;
    
    ret = cblas_sdot(size,
		     x_mem, x_inc,
		     y_mem, y_inc);
#ifdef USE_CUDA
  }
#endif
  return ret;
}


float doSnrm2(unsigned int n,
	      const FloatGPUMirroredMemoryBlock *x,
	      unsigned int inc,
	      unsigned int shift,
	      bool use_gpu) {
  float result;
  const float *x_mem;
#ifdef USE_CUDA
  if (use_gpu) {
    hipblasStatus_t status;
    hipblasHandle_t handle = GPUHelper::getHandler();
    x_mem  = x->getGPUForRead() + shift;
    status = hipblasSetStream(handle, GPUHelper::getCurrentStream());
    checkCublasError(status);
    status = hipblasSnrm2(handle, n, x_mem, inc, &result);
    checkCublasError(status);
  }
  else {
#endif
    x_mem = x->getPPALForRead() + shift;
    result = cblas_snrm2(n, x_mem, inc);
#ifdef USE_CUDA
  }
#endif
  return result;
}

void doSsbmv(CBLAS_ORDER major_type,
	     CBLAS_UPLO uplo,
	     int n, int k,
	     float alpha, FloatGPUMirroredMemoryBlock *a, unsigned int a_lda,
	     FloatGPUMirroredMemoryBlock *x, unsigned int x_inc,
	     float beta, FloatGPUMirroredMemoryBlock *y, unsigned int y_inc,
	     unsigned int a_shift, unsigned int x_shift, unsigned int y_shift,
	     bool use_gpu) {
  const float *a_mem, *x_mem;
  float *y_mem;
#ifdef USE_CUDA
  if (use_gpu) {
    hipblasStatus_t status;
    hipblasHandle_t handle = GPUHelper::getHandler();
    assert(major_type == CblasColMajor);
    a_mem = a->getGPUForRead() + a_shift;
    x_mem = x->getGPUForRead() + x_shift;
    y_mem = y->getGPUForReadAndWrite() + y_shift;

    status = hipblasSetStream(handle, GPUHelper::getCurrentStream());
    checkCublasError(status);
    hipblasFillMode_t uplo_cublas = HIPBLAS_FILL_MODE_UPPER;
    if (uplo == CblasLower) uplo_cublas = HIPBLAS_FILL_MODE_LOWER;
    status = hipblasSsbmv(handle, uplo_cublas,
			 n, k,
			 &alpha, a_mem, a_lda,
			 x_mem, x_inc,
			 &beta, y_mem, y_inc);
    checkCublasError(status);
  }
  else {
#endif
    a_mem = a->getPPALForRead() + a_shift;
    x_mem = x->getPPALForRead() + x_shift;
    y_mem = y->getPPALForReadAndWrite() + y_shift;

    cblas_ssbmv(major_type, uplo,
		n, k,
		alpha, a_mem, a_lda,
		x_mem, x_inc,
		beta, y_mem, y_inc);
#ifdef USE_CUDA
  }
#endif
}

void doClamp(unsigned int N,
	     FloatGPUMirroredMemoryBlock *v,
	     unsigned int stride,
	     unsigned int shift,
	     float lower,
	     float upper,
	     bool use_gpu) {
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    clampKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride, lower, upper);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride)
      *v_mem = april_utils::clamp(*v_mem,lower,upper);
#ifdef USE_CUDA
  }
#endif
}

void doFill(unsigned int N,
	    FloatGPUMirroredMemoryBlock *v,
	    unsigned int stride,
	    unsigned int shift,
	    float value,
	    bool use_gpu) {
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    fillKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride, value);
  }
  else {
#endif
    float *v_mem = v->getPPALForWrite() + shift;
    VECTOR_SSET(N, value, v_mem, stride);
#ifdef USE_CUDA
  }
#endif
}

float doSum(unsigned int N,
	    const FloatGPUMirroredMemoryBlock *v,
	    unsigned int stride,
	    unsigned int shift,
	    bool use_gpu) {
  float sum;
#ifdef USE_CUDA
  if (use_gpu) {
    FloatGPUMirroredMemoryBlock sums(N);
    const float *v_ptr           = v->getGPUForRead() + shift;
    float *sums_ptr              = sums.getGPUForWrite();
    unsigned int units_top       = ceilingPowerOfTwo(N);
    unsigned int top_reduction   = units_top;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    sumVectorFirstReduction<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr,
       sums_ptr,
       top_reduction,
       N, stride);
    for (top_reduction >>= 1; top_reduction != 1; top_reduction >>= 1) {
      computeBlockAndGridSizesForAnArray(top_reduction, block, grid);
      sumVectorNextReduction<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
        (sums_ptr,
         top_reduction,
	 N, stride);
    }
    sum = sums_ptr[0];
  }
  else {
#endif
    const float *v_mem = v->getPPALForRead() + shift;
    sum = 0.0f;
    for (unsigned int i=0; i<N; ++i, v_mem+=stride) sum += *v_mem;
#ifdef USE_CUDA
  }
#endif
  return sum;
}

void doScalarAdd(unsigned int N,
		 FloatGPUMirroredMemoryBlock *v,
		 unsigned int stride,
		 unsigned int shift,
		 float value,
		 bool use_gpu) {
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    scalarAddKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride, value);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem+=stride) *v_mem = *v_mem + value;
#ifdef USE_CUDA
  }
#endif
}

bool doEquals(unsigned int N,
	      const FloatGPUMirroredMemoryBlock *v1,
	      const FloatGPUMirroredMemoryBlock *v2,
	      unsigned int stride1,
	      unsigned int stride2,
	      unsigned int shift1,
	      unsigned int shift2,
	      float epsilon,
	      bool use_gpu) {
  bool eq = true;
#ifdef USE_CUDA
  if (use_gpu) {
    // TODO:
    ERROR_PRINT("CUDA VERSION NOT IMPLEMENTED\n");
  }
  // else {
#endif
    const float *v1_mem = v1->getPPALForRead() + shift1;
    const float *v2_mem = v2->getPPALForRead() + shift2;
    for (unsigned int i=0; i<N && eq; ++i, v1_mem+=stride1, v2_mem+=stride2)
      eq = eq && ((fabsf(*v1_mem - *v2_mem) < epsilon));
#ifdef USE_CUDA
  //  }
#endif
  return eq;
}

void doPLogP(unsigned int N,
	     FloatGPUMirroredMemoryBlock *v,
	     unsigned int stride,
	     unsigned int shift,
	     bool use_gpu) {
#ifdef USE_CUDA
  if (use_gpu) {
    ERROR_PRINT("CUDA VERSION NOT IMPLEMENTED\n");
  }
  //  else {
#endif
  float *v_mem = v->getPPALForReadAndWrite() + shift;
  for (unsigned int i=0; i<N; ++i, v_mem += stride)
    if (*v_mem > 0.0f || *v_mem < 0.0f) *v_mem = (*v_mem) * logf(*v_mem);
#ifdef USE_CUDA
    //  }
#endif  
}

void doLog(unsigned int N,
	   FloatGPUMirroredMemoryBlock *v,
	   unsigned int stride,
	   unsigned int shift,
	   bool use_gpu) {
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    logfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = logf(*v_mem);
#ifdef USE_CUDA
  }
#endif  
}

void doLog1p(unsigned int N,
	     FloatGPUMirroredMemoryBlock *v,
	     unsigned int stride,
	     unsigned int shift,
	     bool use_gpu) {
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    log1pfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = log1pf(*v_mem);
#ifdef USE_CUDA
  }
#endif  
}

void doExp(unsigned int N,
	   FloatGPUMirroredMemoryBlock *v,
	   unsigned int stride,
	   unsigned int shift,
	   bool use_gpu) {
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    expfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = expf(*v_mem);
#ifdef USE_CUDA
  }
#endif  
}

void doSqrt(unsigned int N,
	    FloatGPUMirroredMemoryBlock *v,
	    unsigned int stride,
	    unsigned int shift,
	    bool use_gpu) {
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    sqrtfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = sqrtf(*v_mem);
#ifdef USE_CUDA
  }
#endif  
}

void doTanh(unsigned int N,
	    FloatGPUMirroredMemoryBlock *v,
	    unsigned int stride,
	    unsigned int shift,
	    bool use_gpu) {
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    tanhfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride) *v_mem = tanhf(*v_mem);
#ifdef USE_CUDA
  }
#endif
}

void doPow(unsigned int N,
	   FloatGPUMirroredMemoryBlock *v,
	   unsigned int stride,
	   unsigned int shift,
	   float value,
	   bool use_gpu) {
#ifdef USE_CUDA
  if (use_gpu) {
    float *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    powfFuncKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride, value);
  }
  else {
#endif
    float *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem += stride)
      *v_mem = powf(*v_mem, value);
#ifdef USE_CUDA
  }
#endif
}
