#include "hip/hip_runtime.h"
/*
 * This file is part of the Neural Network modules of the APRIL toolkit (A
 * Pattern Recognizer In Lua).
 *
 * Copyright 2012, Salvador España-Boquera, Adrian Palacios Corella, Francisco
 * Zamora-Martinez
 *
 * The APRIL-MLP toolkit is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License version 3 as
 * published by the Free Software Foundation
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License
 * for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this library; if not, write to the Free Software Foundation,
 * Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA
 *
 */
#include "wrapper.h"
#include "cuda_utils.h"
#include "unused_variable.h"

#ifdef USE_CUDA
/***************************************
 ************** CUDA SECTION ***********
 ***************************************/

template<typename T>
__global__ void scalarAddKernel(T *v, unsigned int N, unsigned int stride,
				T value) {
  unsigned int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (x_idx < N) {
    T *aux = v + x_idx*stride;
    *aux += value;
  }
}

#endif

/***************************************
 *********** TEMPLATE SECTION **********
 ***************************************/

template<typename T>
void doScalarAdd(unsigned int N,
		 GPUMirroredMemoryBlock<T> *v,
		 unsigned int stride,
		 unsigned int shift,
		 T value,
		 bool use_gpu) {
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    T *v_ptr = v->getGPUForReadAndWrite() + shift;
    dim3 block, grid;
    computeBlockAndGridSizesForAnArray(N, block, grid);
    scalarAddKernel<<<grid, block, 0, GPUHelper::getCurrentStream()>>>
      (v_ptr, N, stride, value);
  }
  else {
#endif
    T *v_mem = v->getPPALForReadAndWrite() + shift;
    for (unsigned int i=0; i<N; ++i, v_mem+=stride) *v_mem = *v_mem + value;
#ifdef USE_CUDA
  }
#endif
}

template void doScalarAdd<float>(unsigned int N,
				 GPUMirroredMemoryBlock<float> *v,
				 unsigned int stride,
				 unsigned int shift,
				 float value,
				 bool use_gpu);

template void doScalarAdd<ComplexF>(unsigned int N,
				    GPUMirroredMemoryBlock<ComplexF> *v,
				    unsigned int stride,
				    unsigned int shift,
				    ComplexF value,
				    bool use_gpu);
